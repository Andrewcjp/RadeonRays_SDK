#include "hip/hip_runtime.h"
#ifndef MATH_UTILS_CU
#define MATH_UTILS_CU

inline __host__ __device__ float native_recip(float in)
{
    return 1.f/in;
}

inline __host__ __device__ float3 native_recip(float3 in)
{
    return make_float3( native_recip(in.x), 
                        native_recip(in.y), 
                        native_recip(in.z));
}

inline __host__ __device__ float mad(float a, float b, float c)
{
    return fmaf(a,b,c);
}

inline __host__ __device__ float3 mad(float3 a, float3 b, float3 c)
{
    return  make_float3(fmaf(a.x, b.x, c.x), 
                        fmaf(a.y, b.y, c.y), 
                        fmaf(a.z, b.z, c.z));

}

inline __host__ __device__ float3 max(float3 a, float3 b)
{
    return  make_float3(fmaxf(a.x, b.x), 
                        fmaxf(a.y, b.y), 
                        fmaxf(a.z, b.z));

}

inline __host__ __device__ float3 min(float3 a, float3 b)
{
    return  make_float3(fminf(a.x, b.x), 
                        fminf(a.y, b.y), 
                        fminf(a.z, b.z));

}

//from cuda cutil_math.h

////////////////////////////////////////////////////////////////////////////////
// dot product
////////////////////////////////////////////////////////////////////////////////

inline __host__ __device__ float dot(float2 a, float2 b)
{ 
    return a.x * b.x + a.y * b.y;
}
inline __host__ __device__ float dot(float3 a, float3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}
inline __host__ __device__ float dot(float4 a, float4 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

inline __host__ __device__ int dot(int2 a, int2 b)
{ 
    return a.x * b.x + a.y * b.y;
}
inline __host__ __device__ int dot(int3 a, int3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}
inline __host__ __device__ int dot(int4 a, int4 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

inline __host__ __device__ uint dot(uint2 a, uint2 b)
{ 
    return a.x * b.x + a.y * b.y;
}
inline __host__ __device__ uint dot(uint3 a, uint3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}
inline __host__ __device__ uint dot(uint4 a, uint4 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

////////////////////////////////////////////////////////////////////////////////
// length
////////////////////////////////////////////////////////////////////////////////

inline __host__ __device__ float length(float2 v)
{
    return sqrtf(dot(v, v));
}
inline __host__ __device__ float length(float3 v)
{
    return sqrtf(dot(v, v));
}
inline __host__ __device__ float length(float4 v)
{
    return sqrtf(dot(v, v));
}

////////////////////////////////////////////////////////////////////////////////
// normalize
////////////////////////////////////////////////////////////////////////////////

inline __host__ __device__ float2 normalize(float2 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}
inline __host__ __device__ float3 normalize(float3 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}
inline __host__ __device__ float4 normalize(float4 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}

////////////////////////////////////////////////////////////////////////////////
// cross product
////////////////////////////////////////////////////////////////////////////////

inline __host__ __device__ float3 cross(float3 a, float3 b)
{ 
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x); 
}

#endif //MATH_UTILS_CU
