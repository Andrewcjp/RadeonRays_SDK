#include "hip/hip_runtime.h"
/**********************************************************************
Copyright (c) 2016 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without __restrict__ion, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
********************************************************************/
/**
    \file intersect_hlbvh_stack.cl
    \author Dmitry Kozlov
    \version 1.0
    \brief HLBVH build implementation

    IntersectorHlbvh implementation is based on the following paper:
    "HLBVH: Hierarchical LBVH Construction for Real-Time Ray Tracing"
    Jacopo Pantaleoni (NVIDIA), David Luebke (NVIDIA), in High Performance Graphics 2010, June 2010
    https://research.nvidia.com/sites/default/files/publications/HLBVH-final.pdf

    Pros:
        -Very fast to build and update.
    Cons:
        -Poor BVH quality, slow traversal.
 */

 /*************************************************************************
  INCLUDES
  **************************************************************************/
#include "common.cu"

 /*************************************************************************
   EXTENSIONS
**************************************************************************/



/*************************************************************************
TYPE DEFINITIONS
**************************************************************************/
#define STARTIDX(x)     (((int)((x).child0)))
#define LEAFNODE(x)     (((x).child0) == ((x).child1))
#define GLOBAL_STACK_SIZE 32
#define SHORT_STACK_SIZE 16
#define WAVEFRONT_SIZE 64

typedef struct bvh_node
{
    int parent;
    int child0;
    int child1;
    int next;
    __device__ ~bvh_node(){};
};

typedef struct Face
{
    // Vertex indices
    int idx[3];
    // Shape maks
    int shape_mask;
    // Shape ID
    int shape_id;
    // Primitive ID
    int prim_id;
    __device__ ~Face(){};
};

__global__
void occluded_main(
    hipLaunchParm lp,
    // Bvh nodes
    bvh_node const * __restrict__ nodes,
    // Bounding boxes
    bbox const* __restrict__ bounds,
    // Triangles vertices
    float3 const * __restrict__ vertices,
    // Triangle indices
    Face const* faces,
    // Rays
    ray const * __restrict__ rays,
    // Number of rays in rays buffer
    int const * __restrict__ num_rays,
    // Stack memory
    int* stack,
    // Hit results: 1 for hit and -1 for miss
    int* hits
    )
{
    int global_id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    int local_id = hipBlockIdx_x;
    int group_id = hipBlockDim_x;

    // Handle only working set
    if (global_id < *num_rays)
    {
        ray const r = rays[global_id];

        if (ray_is_active(&r))
        {
            // Allocate stack in memory 
            int* gm_stack_base = stack + (group_id * WAVEFRONT_SIZE + local_id) * GLOBAL_STACK_SIZE;
            int* gm_stack = gm_stack_base;
            // Allocate stack in LDS
            int lds[SHORT_STACK_SIZE * WAVEFRONT_SIZE];
            int* lm_stack_base = lds + local_id;
            int* lm_stack = lm_stack_base;

            float3 const ray_origin = make_float3(r.o.x, r.o.y, r.o.z);
            float3 const ray_dir = make_float3(r.d.x, r.d.y, r.d.z);

            // Precompute inverse direction and origin / dir for bbox testing
            float3 const invdir = safe_invdir(r);
            float3 const oxinvdir = -1 * ray_origin * invdir;
            // Intersection parametric distance
            float const t_max = r.o.w;

            // Current node address
            int addr = 0;
            // Current closest intersection leaf index
            int isect_idx = INVALID_IDX;

            //  Initalize local stack
            *lm_stack = INVALID_IDX;
            lm_stack += WAVEFRONT_SIZE;

            // Start from 0 node (root)
            while (addr != INVALID_IDX)
            {
                // Fetch next node
                bvh_node const node = nodes[addr];

                // Check if it is a leaf
                if (LEAFNODE(node))
                {
                    Face face = faces[STARTIDX(node)];
                    // Leafs directly store vertex indices
                    // so we load vertices directly
                    float3 const v1 = vertices[face.idx[0]];
                    float3 const v2 = vertices[face.idx[1]];
                    float3 const v3 = vertices[face.idx[2]];
                    // Intersect triangle
                    float const f = fast_intersect_triangle(r, v1, v2, v3, t_max);
                    // If hit update closest hit distance and index
                    if (f < t_max)
                    {
                        hits[global_id] = HIT_MARKER;
                        return;
                    }
                }
                else
                {
                    // It is internal node, so intersect vs both children bounds
                    float2 const s0 = fast_intersect_bbox1(bounds[node.child0], invdir, oxinvdir, t_max);
                    float2 const s1 = fast_intersect_bbox1(bounds[node.child1], invdir, oxinvdir, t_max);

                    // Determine which one to traverse
                    bool const traverse_c0 = (s0.x <= s0.y);
                    bool const traverse_c1 = (s1.x <= s1.y);
                    bool const c1first = traverse_c1 && (s0.x > s1.x);

                    if (traverse_c0 || traverse_c1)
                    {
                        int deferred = -1;

                        // Determine which one to traverse first
                        if (c1first || !traverse_c0)
                        {
                            // Right one is closer or left one not travesed
                            addr = node.child1;
                            deferred = node.child0;
                        }
                        else
                        {
                            // Traverse left node otherwise
                            addr = node.child0;
                            deferred = node.child1;
                        }

                        // If we traverse both children we need to postpone the node
                        if (traverse_c0 && traverse_c1)
                        {
                            // If short stack is full, we offload it into memory
                            if (lm_stack - lm_stack_base >= SHORT_STACK_SIZE * WAVEFRONT_SIZE)
                            {
                                for (int i = 1; i < SHORT_STACK_SIZE; ++i)
                                {
                                    gm_stack[i] = lm_stack_base[i * WAVEFRONT_SIZE];
                                }

                                gm_stack += SHORT_STACK_SIZE;
                                lm_stack = lm_stack_base + WAVEFRONT_SIZE;
                            }

                            *lm_stack = deferred;
                            lm_stack += WAVEFRONT_SIZE;
                        }

                        // Continue traversal
                        continue;
                    }
                }

                // Try popping from local stack
                lm_stack -= WAVEFRONT_SIZE;
                addr = *(lm_stack);

                // If we popped INVALID_IDX then check stack
                if (addr == INVALID_IDX && gm_stack > gm_stack_base)
                {
                    // Adjust stack pointer
                    gm_stack -= SHORT_STACK_SIZE;
                    // Copy data from memory to LDS
                    for (int i = 1; i < SHORT_STACK_SIZE; ++i)
                    {
                        lm_stack_base[i * WAVEFRONT_SIZE] = gm_stack[i];
                    }
                    // Point local stack pointer to the end
                    lm_stack = lm_stack_base + (SHORT_STACK_SIZE - 1) * WAVEFRONT_SIZE;
                    addr = lm_stack_base[WAVEFRONT_SIZE * (SHORT_STACK_SIZE - 1)];
                }
            }

            // Finished traversal, but no intersection found
            hits[global_id] = MISS_MARKER;
        }
    }
}

__global__
void intersect_main(
    hipLaunchParm lp,
    // Bvh nodes
    bvh_node const* __restrict__ nodes,
    // Bounding boxes
    bbox const* __restrict__ bounds,
    // Triangles vertices
    float3 const* __restrict__ vertices,
    // Faces
    Face const* __restrict__ faces,
    // Rays
    ray const* __restrict__ rays,
    // Number of rays in rays buffer
    int const* __restrict__ num_rays,
    // Stack memory
    int* stack,
    // Hit data
    Intersection* hits)
{
    int global_id = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    int local_id = hipBlockIdx_x;
    int group_id = hipBlockDim_x;

    // Handle only working subset
    if (global_id < *num_rays)
    {
        ray const r = rays[global_id];

        if (ray_is_active(&r))
        {
            // Allocate stack in memory 
            int* gm_stack_base = stack + (group_id * WAVEFRONT_SIZE + local_id) * GLOBAL_STACK_SIZE;
            int* gm_stack = gm_stack_base;
            // Allocate stack in LDS
            int lds[SHORT_STACK_SIZE * WAVEFRONT_SIZE];
            int* lm_stack_base = lds + local_id;
            int* lm_stack = lm_stack_base;

            float3 const ray_origin = make_float3(r.o.x, r.o.y, r.o.z);
            float3 const ray_dir = make_float3(r.d.x, r.d.y, r.d.z);

            // Precompute inverse direction and origin / dir for bbox testing
            float3 const invdir = safe_invdir(r);
            float3 const oxinvdir = -1 * ray_origin * invdir;
            // Intersection parametric distance
            float t_max = r.o.w;

            // Current node address
            int addr = 0;
            // Current closest intersection leaf index
            int isect_idx = INVALID_IDX;

            //  Initalize local stack
            *lm_stack = INVALID_IDX;
            lm_stack += WAVEFRONT_SIZE;

            // Start from 0 node (root)
            while (addr != INVALID_IDX)
            {
                // Fetch next node
                bvh_node const node = nodes[addr];

                // Check if it is a leaf
                if (LEAFNODE(node))
                {
                    Face face = faces[STARTIDX(node)];
                    // Leafs directly store vertex indices
                    // so we load vertices directly
                    float3 const v1 = vertices[face.idx[0]];
                    float3 const v2 = vertices[face.idx[1]];
                    float3 const v3 = vertices[face.idx[2]];
                    // Intersect triangle
                    float const f = fast_intersect_triangle(r, v1, v2, v3, t_max);
                    // If hit update closest hit distance and index
                    if (f < t_max)
                    {
                        t_max = f;
                        isect_idx = STARTIDX(node);
                    }
                }
                else
                {
                    // It is internal node, so intersect vs both children bounds
                    float2 const s0 = fast_intersect_bbox1(bounds[node.child0], invdir, oxinvdir, t_max);
                    float2 const s1 = fast_intersect_bbox1(bounds[node.child1], invdir, oxinvdir, t_max);

                    // Determine which one to traverse
                    bool const traverse_c0 = (s0.x <= s0.y);
                    bool const traverse_c1 = (s1.x <= s1.y);
                    bool const c1first = traverse_c1 && (s0.x > s1.x);

                    if (traverse_c0 || traverse_c1)
                    {
                        int deferred = -1;

                        // Determine which one to traverse first
                        if (c1first || !traverse_c0)
                        {
                            // Right one is closer or left one not travesed
                            addr = node.child1;
                            deferred = node.child0;
                        }
                        else
                        {
                            // Traverse left node otherwise
                            addr = node.child0;
                            deferred = node.child1;
                        }

                        // If we traverse both children we need to postpone the node
                        if (traverse_c0 && traverse_c1)
                        {
                            // If short stack is full, we offload it into memory
                            if ( lm_stack - lm_stack_base >= SHORT_STACK_SIZE * WAVEFRONT_SIZE)
                            {
                                for (int i = 1; i < SHORT_STACK_SIZE; ++i)
                                {
                                    gm_stack[i] = lm_stack_base[i * WAVEFRONT_SIZE];
                                }

                                gm_stack += SHORT_STACK_SIZE;
                                lm_stack = lm_stack_base + WAVEFRONT_SIZE;
                            }

                            *lm_stack = deferred;
                            lm_stack += WAVEFRONT_SIZE;
                        }

                        // Continue traversal
                        continue;
                    }
                }

                // Try popping from local stack
                lm_stack -= WAVEFRONT_SIZE;
                addr = *(lm_stack);

                // If we popped INVALID_IDX then check stack
                if (addr == INVALID_IDX && gm_stack > gm_stack_base)
                {
                    // Adjust stack pointer
                    gm_stack -= SHORT_STACK_SIZE;
                    // Copy data from memory to LDS
                    for (int i = 1; i < SHORT_STACK_SIZE; ++i)
                    {
                        lm_stack_base[i * WAVEFRONT_SIZE] = gm_stack[i];
                    }
                    // Point local stack pointer to the end
                    lm_stack = lm_stack_base + (SHORT_STACK_SIZE - 1) * WAVEFRONT_SIZE;
                    addr = lm_stack_base[WAVEFRONT_SIZE * (SHORT_STACK_SIZE - 1)];
                }
            }

            // Check if we have found an intersection
            if (isect_idx != INVALID_IDX)
            {
                // Fetch the node & vertices
                Face const face = faces[isect_idx];
                float3 const v1 = vertices[face.idx[0]];
                float3 const v2 = vertices[face.idx[1]];
                float3 const v3 = vertices[face.idx[2]];
                // Calculate hit position
                float3 const p = ray_origin + ray_dir * t_max;
                // Calculte barycentric coordinates
                float2 const uv = triangle_calculate_barycentrics(p, v1, v2, v3);
                // Update hit information
                hits[global_id].shape_id = face.shape_id;
                hits[global_id].prim_id = face.prim_id;
                hits[global_id].uvwt = make_float4(uv.x, uv.y, 0.f, t_max);
            }
            else
            {
                // Miss here
                hits[global_id].shape_id = MISS_MARKER;
                hits[global_id].prim_id = MISS_MARKER;
            }
        }
    }
}



